
#include <hip/hip_runtime.h>
/***************************************************************************//**
**  \mainpage Monte Carlo eXtreme - GPU accelerated Monte Carlo Photon Migration
**
**  \author Qianqian Fang <q.fang at neu.edu>
**  \copyright Qianqian Fang, 2009-2024
**
**  \section sref Reference
**  \li \c (\b Fang2009) Qianqian Fang and David A. Boas,
**          <a href="http://www.opticsinfobase.org/abstract.cfm?uri=oe-17-22-20178">
**          "Monte Carlo Simulation of Photon Migration in 3D Turbid Media Accelerated
**          by Graphics Processing Units,"</a> Optics Express, 17(22) 20178-20190 (2009).
**  \li \c (\b Yu2018) Leiming Yu, Fanny Nina-Paravecino, David Kaeli, and Qianqian Fang,
**          "Scalable and massively parallel Monte Carlo photon transport
**           simulations for heterogeneous computing platforms," J. Biomed. Optics,
**           23(1), 010504, 2018. https://doi.org/10.1117/1.JBO.23.1.010504
**  \li \c (\b Yan2020) Shijie Yan and Qianqian Fang* (2020), "Hybrid mesh and voxel
**          based Monte Carlo algorithm for accurate and efficient photon transport
**          modeling in complex bio-tissues," Biomed. Opt. Express, 11(11)
**          pp. 6262-6270. https://doi.org/10.1364/BOE.409468
**
**  \section sformat Formatting
**          Please always run "make pretty" inside the \c src folder before each commit.
**          The above command requires \c astyle to perform automatic formatting.
**
**  \section slicense License
**          GPL v3, see LICENSE.txt for details
*******************************************************************************/

/***************************************************************************//**
\file    mcx_vector_math.cu

@brief    Common math operations on vector types.
*******************************************************************************/

/**
 * @brief Adding two float3 vectors c=a+b
 */

inline __device__ float3 operator +(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

/**
 * @brief Increatment a float3 vector by another float3, a+=b
 */

inline __device__ void operator +=(float3& a, const float3& b) {
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

/**
 * @brief Subtracting two float3 vectors c=a+b
 */

inline __device__ float3 operator -(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}


/**
 * @brief Negating a float3 vector c=-a
 */

inline __device__ float3 operator -(const float3& a) {
    return make_float3(-a.x, -a.y, -a.z);
}

/**
 * @brief Front-multiplying a float3 with a scalar c=a*b
 */

inline __device__ float3 operator *(const float& a, const float3& b) {
    return make_float3(a * b.x, a * b.y, a * b.z);
}

/**
 * @brief Post-multiplying a float3 with a scalar c=a*b
 */

inline __device__ float3 operator *(const float3& a, const float& b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

/**
 * @brief Multiplying two float3 vectors c=a*b
 */

inline __device__ float3 operator *(const float3& a, const float3& b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

/**
 * @brief Dot-product of two float3 vectors c=a*b
 */

inline __device__ float dot(const float3& a, const float3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

/**
 * @brief Adding two uint3 vectors c=a+b
 */

inline __device__ uint3 operator +(const uint3& a, const uint3& b) {
    return make_uint3(a.x + b.x, a.y + b.y, a.z + b.z);
}

/**
 * @brief Cast uint3 to float3
 */

inline __device__ float3 make_float3(const uint3& a) {
    return make_float3(float(a.x), float(a.y), float(a.z));
}


/**
 * @brief Cross-product of two float3 vectors c=axb
 */

inline __device__ float3 cross(const float3& a, const float3& b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

/**
 * @brief Length of a float3 vector
 */

inline __device__ float length(const float3& v) {
    return sqrtf(dot(v, v));
}

/**
 * @brief Division between a float3 vector and a float
 */

inline __device__ float3 operator/(const float3& a, const float& b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

/**
 * @brief Divide a float3 vector by a float
 */

inline __device__ void operator/=(float3& a, const float& b) {
    a.x /= b;
    a.y /= b;
    a.z /= b;
}