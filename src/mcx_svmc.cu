#include "hip/hip_runtime.h"
/***************************************************************************//**
**  \mainpage Monte Carlo eXtreme - GPU accelerated Monte Carlo Photon Migration
**
**  \author Qianqian Fang <q.fang at neu.edu>
**  \copyright Qianqian Fang, 2009-2024
**
**  \section sref Reference
**  \li \c (\b Fang2009) Qianqian Fang and David A. Boas,
**          <a href="http://www.opticsinfobase.org/abstract.cfm?uri=oe-17-22-20178">
**          "Monte Carlo Simulation of Photon Migration in 3D Turbid Media Accelerated
**          by Graphics Processing Units,"</a> Optics Express, 17(22) 20178-20190 (2009).
**  \li \c (\b Yu2018) Leiming Yu, Fanny Nina-Paravecino, David Kaeli, and Qianqian Fang,
**          "Scalable and massively parallel Monte Carlo photon transport
**           simulations for heterogeneous computing platforms," J. Biomed. Optics,
**           23(1), 010504, 2018. https://doi.org/10.1117/1.JBO.23.1.010504
**  \li \c (\b Yan2020) Shijie Yan and Qianqian Fang* (2020), "Hybrid mesh and voxel
**          based Monte Carlo algorithm for accurate and efficient photon transport
**          modeling in complex bio-tissues," Biomed. Opt. Express, 11(11)
**          pp. 6262-6270. https://doi.org/10.1364/BOE.409468
**
**  \section sformat Formatting
**          Please always run "make pretty" inside the \c src folder before each commit.
**          The above command requires \c astyle to perform automatic formatting.
**
**  \section slicense License
**          GPL v3, see LICENSE.txt for details
*******************************************************************************/

/***************************************************************************//**
\file    mcx_svmc.cu

@brief    GPU kernel for volume preprocessing for SVMC simulations(Yan2020).
*******************************************************************************/

#include "mcx_svmc.h"

#include <stdint.h>

#include "mcx_tictoc.h"
#include "mcx_const.h"
#include "mcx_vector_math.cu"

// host function signatures
void pad_replicate_volume(unsigned int* vol, unsigned int** new_vol, unsigned int pad_size, unsigned int dimx,
                          unsigned int dimy, unsigned int dimz);
void gaussian_filter(float** filter, unsigned int sizex, unsigned int sizey, unsigned int sizez, float std);
void mcx_cu_assess(hipError_t cuerr, const char* file, const int linenum);

// device function signatures
__global__ void init_lower_label(unsigned char* vol_new, unsigned int* vol);
__global__ void create_binary_mask(unsigned int* vol, float* binary_mask, unsigned int label);
__global__ void gaussian_blur(float* binary_mask, float* mask);
__global__ void split_voxel(float* scalar_field, unsigned char* new_vol, unsigned int label);
__device__ float3 interpolate(float3 a, float3 b, float a_val, float b_val, float isovalue);
__device__ unsigned int flatten_3d_to_1d(uint3 idx3d, uint3 dim);

/**
 * @brief      macro to report CUDA errors
 */
#define CUDA_ASSERT(a)      mcx_cu_assess((a),__FILE__,__LINE__)

/**
 * gaussian filter parameters
 */
#define MCX_SVMC_GKERNEL_SIZE 3U    // must be an odd positive integer
#define MCX_SVMC_GKERNEL_STD  1.0f
#define MCX_SVMC_ISOVALUE     0.5f

/**
 * Indices of the vertices in the local coordinate system
 */
__constant__ uint3 cube_vertices_local[8] = {
    {0, 0, 0}, {1, 0, 0}, {1, 0, 1}, {0, 0, 1},
    {0, 1, 0}, {1, 1, 0}, {1, 1, 1}, {0, 1, 1}
};

/**
 * gaussian filter for smoothing binary volume
 */
__constant__ float gfilter[MCX_SVMC_GKERNEL_SIZE * MCX_SVMC_GKERNEL_SIZE * MCX_SVMC_GKERNEL_SIZE];


/**
 * edge[i] joins edge_vertices[i][0] and edge_vertices[i][1]
 * adapted from https://paulbourke.net/geometry/polygonise/
 */
__constant__ uint8_t edge_vertices[12][2] = {
    {0, 1}, {1, 2}, {2, 3}, {0, 3}, {4, 5}, {5, 6}, {6, 7}, {4, 7}, {0, 4}, {1, 5}, {2, 6}, {3, 7}
};

/**
 * (edge_intersections[i] >> j) & 1 is 1 if isosurface intersects edge[j], where 0 <= j <= 11
 * adapted from https://paulbourke.net/geometry/polygonise/
 */
__constant__ uint16_t edge_intersections[256] = {
    0x000, 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
    0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
    0x190, 0x099, 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
    0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
    0x230, 0x339, 0x033, 0x13a, 0x636, 0x73f, 0x435, 0x53c,
    0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
    0x3a0, 0x2a9, 0x1a3, 0x0aa, 0x7a6, 0x6af, 0x5a5, 0x4ac,
    0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
    0x460, 0x569, 0x663, 0x76a, 0x066, 0x16f, 0x265, 0x36c,
    0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
    0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0x0ff, 0x3f5, 0x2fc,
    0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
    0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x055, 0x15c,
    0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
    0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0x0cc,
    0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
    0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
    0x0cc, 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
    0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
    0x15c, 0x055, 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
    0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
    0x2fc, 0x3f5, 0x0ff, 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
    0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
    0x36c, 0x265, 0x16f, 0x066, 0x76a, 0x663, 0x569, 0x460,
    0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
    0x4ac, 0x5a5, 0x6af, 0x7a6, 0x0aa, 0x1a3, 0x2a9, 0x3a0,
    0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
    0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x033, 0x339, 0x230,
    0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
    0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x099, 0x190,
    0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
    0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x000
};

/**
 * In ith (0-255) configuration, triangle_vertices[i][j*3], triangle_vertices[i][j*3+1]
 * and triangle_vertices[i][j*3+2] are vertices of the jth triangle
 * adapted from https://paulbourke.net/geometry/polygonise/
 */
__constant__ int8_t triangle_vertices[256][16] = {
    {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
    {3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
    {3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
    {3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
    {9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
    {2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
    {8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
    {4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
    {3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
    {1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
    {4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
    {4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
    {5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
    {2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
    {9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
    {0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
    {2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
    {10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
    {5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
    {5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
    {9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
    {1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
    {10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
    {8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
    {2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
    {7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
    {2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
    {11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
    {5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
    {11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
    {11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
    {9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
    {2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
    {6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
    {3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
    {6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
    {10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
    {6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
    {8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
    {7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
    {3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
    {5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
    {0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
    {9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
    {8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
    {5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
    {0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
    {6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
    {10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
    {10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
    {8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
    {1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
    {0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
    {10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
    {3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
    {6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
    {9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
    {8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
    {3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
    {6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
    {0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
    {10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
    {10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
    {2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
    {7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
    {7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
    {2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
    {1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
    {11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
    {8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
    {0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
    {7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
    {10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
    {2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
    {6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
    {7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
    {2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
    {1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
    {10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
    {10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
    {0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
    {7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
    {6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
    {8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
    {9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
    {6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
    {4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
    {10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
    {8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
    {0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
    {1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
    {8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
    {10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
    {4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
    {10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
    {5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
    {11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
    {9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
    {6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
    {7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
    {3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
    {7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
    {3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
    {6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
    {9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
    {1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
    {4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
    {7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
    {6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
    {3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
    {0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
    {6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
    {0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
    {11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
    {6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
    {5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
    {9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
    {1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
    {1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
    {10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
    {0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
    {5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
    {10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
    {11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
    {9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
    {7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
    {2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
    {8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
    {9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
    {9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
    {1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
    {9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
    {9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
    {5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
    {0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
    {10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
    {2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
    {0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
    {0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
    {9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
    {5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
    {3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
    {5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
    {8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
    {0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
    {9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
    {0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
    {1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
    {3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
    {4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
    {9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
    {11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
    {11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
    {2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
    {9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
    {3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
    {1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
    {4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
    {4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
    {0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
    {3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
    {3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
    {0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
    {9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
    {1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
    {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}
};

/**
 * @brief      Preprocess media volume for SVMC simulation
 *
 * @param      cfg   Simulation Configuration
 * @param      gpu   GPU info
 */
void mcx_svmc_preprocess(Config* cfg, GPUInfo* gpu) {
    // if volume is not an 3D integer array, do nothing
    if (cfg->mediabyte > 4) {
        return;
    }

    // start timer
    MCX_FPRINTF(cfg->flog, "Preprocessing volume for SVMC simulation... \t");
    unsigned int tic = StartTimer();

    // activate a GPU
    int gpuid = cfg->deviceid[0] - 1;
    CUDA_ASSERT(hipSetDevice(gpuid));

    // dimension of the volume and the padded volume
    uint3 vol_dim = cfg->dim;
    unsigned int pad_size = MCX_SVMC_GKERNEL_SIZE / 2;
    uint3 vol_padded_dim = make_uint3(vol_dim.x + 2 * pad_size, vol_dim.y + 2 * pad_size, vol_dim.z + 2 * pad_size);
    unsigned long long vol_length = vol_dim.x * vol_dim.y * vol_dim.z;
    unsigned long long vol_padded_length = vol_padded_dim.x * vol_padded_dim.y * vol_padded_dim.z;

    // pad 3D volume for filtering
    unsigned int* vol_padded = NULL;
    pad_replicate_volume(cfg->vol, &vol_padded, pad_size, vol_dim.x, vol_dim.y, vol_dim.z);

    // upload padded 3D volume to GPU
    unsigned int* gvol_padded = NULL;
    CUDA_ASSERT(hipMalloc((void**)&gvol_padded, sizeof(unsigned int) * vol_padded_length));
    CUDA_ASSERT(hipMemcpy(gvol_padded, vol_padded, sizeof(unsigned int) * vol_padded_length, hipMemcpyHostToDevice));

    // generate gaussian filter and upload it to constant memory
    float* filter = NULL;
    gaussian_filter(&filter, MCX_SVMC_GKERNEL_SIZE, MCX_SVMC_GKERNEL_SIZE, MCX_SVMC_GKERNEL_SIZE, MCX_SVMC_GKERNEL_STD);
    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gfilter), filter,
                                   sizeof(float) * MCX_SVMC_GKERNEL_SIZE * MCX_SVMC_GKERNEL_SIZE * MCX_SVMC_GKERNEL_SIZE,
                                   0, hipMemcpyHostToDevice));

    // allocate global memory buffers
    float* gbinary_mask = NULL;
    float* gmask = NULL;
    unsigned int* gvol = NULL;
    unsigned char* gvol_new = NULL;
    CUDA_ASSERT(hipMalloc((void**)&gbinary_mask, sizeof(float) * vol_padded_length));
    CUDA_ASSERT(hipMalloc((void**)&gmask, sizeof(float) * vol_length));
    CUDA_ASSERT(hipMalloc((void**)&gvol, sizeof(unsigned int) * vol_length));
    CUDA_ASSERT(hipMalloc((void**)&gvol_new, sizeof(unsigned int) * vol_length * 2));

    // copy old vol
    CUDA_ASSERT(hipMemcpy(gvol, cfg->vol, sizeof(unsigned int) * vol_length, hipMemcpyHostToDevice));

    // init new vol to 0
    CUDA_ASSERT(hipMemset((void*)gvol_new, 0, sizeof(unsigned int) * vol_length * 2));

    // iterate over all medium labels
    dim3 grid, block;

    // init lower label of the gvol_new
    grid = dim3(vol_dim.x, vol_dim.y, vol_dim.z);
    block = dim3(1, 1, 1);
    init_lower_label <<< grid, block>>>(gvol_new, gvol);
    hipDeviceSynchronize();

    for (unsigned int label = 0; label < cfg->medianum; ++label) {
        // generate binary mask for each label
        grid = dim3(vol_padded_dim.x, vol_padded_dim.y, vol_padded_dim.z);
        block = dim3(1, 1, 1);
        create_binary_mask <<< grid, block>>>(gvol_padded, gbinary_mask, label);
        hipDeviceSynchronize();

        // smooth the binary mask using gaussian blur
        grid = dim3(vol_dim.x, vol_dim.y, vol_dim.z);
        block = dim3(1, 1, 1);
        gaussian_blur <<< grid, block>>>(gbinary_mask, gmask);
        hipDeviceSynchronize();

        // generate intra-voxel boundary surface using marching cube algorithm
        grid = dim3(vol_dim.x - 1, vol_dim.y - 1, vol_dim.z - 1);
        block = dim3(1, 1, 1);
        split_voxel <<< grid, block>>>(gmask, gvol_new, label);
        hipDeviceSynchronize();
    }

    // report elapsed time
    MCX_FPRINTF(cfg->flog, "complete:  \t%d ms\n", GetTimeMillis() - tic);

    // download new volume and overwrite the old volume
    cfg->vol = (unsigned int*)malloc(sizeof(unsigned int) * vol_length * 2);
    CUDA_ASSERT(hipMemcpy(cfg->vol, gvol_new, sizeof(unsigned int) * vol_length * 2, hipMemcpyDeviceToHost));

    // enable svmc mode
    cfg->mediabyte = MEDIA_2LABEL_SPLIT;

    // adjust source position to compensate for the grid offset between mcx and svmc
    cfg->srcpos.x += 0.5f;
    cfg->srcpos.y += 0.5f;
    cfg->srcpos.z += 0.5f;

    // TODO: adjust detector position to compensate for the grid offset between mcx and svmc

    // add detector mask
    mcx_maskdet(cfg);

    // clear
    if (vol_padded) {
        free(vol_padded);
    }

    if (filter) {
        free(filter);
    }

    if (gvol_padded) {
        CUDA_ASSERT(hipFree(gvol_padded));
    }

    if (gbinary_mask) {
        CUDA_ASSERT(hipFree(gbinary_mask));
    }

    if (gmask) {
        CUDA_ASSERT(hipFree(gmask));
    }

    if (gvol) {
        CUDA_ASSERT(hipFree(gvol));
    }

    if (gvol_new) {
        CUDA_ASSERT(hipFree(gvol_new));
    }
}

/**
 * @brief      Pad a 3D volume by replicating values
 *
 * @param      vol       The volume
 * @param      new_vol   The new volume
 * @param[in]  pad_size  The pad size
 * @param[in]  dimx      The dimx
 * @param[in]  dimy      The dimy
 * @param[in]  dimz      The dimz
 */
void pad_replicate_volume(unsigned int* vol, unsigned int** new_vol, unsigned int pad_size,
                          unsigned int dimx, unsigned int dimy, unsigned int dimz) {
    unsigned int new_dimx = dimx + pad_size * 2;
    unsigned int new_dimy = dimy + pad_size * 2;
    unsigned int new_dimz = dimz + pad_size * 2;
    *new_vol = (unsigned int*)calloc(new_dimx * new_dimy * new_dimz, sizeof(unsigned int));

    // copy vol values
    for (unsigned int i = 0; i < dimx; ++i) {
        for (unsigned int j = 0; j < dimy; ++j) {
            for (unsigned int k = 0; k < dimz; ++k) {
                (*new_vol)[pad_size + i + (pad_size + j) * new_dimx + (pad_size + k) * new_dimx * new_dimy] =
                    vol[i + j * dimx + k * dimx * dimy];
            }
        }
    }

    // pad along -x
    for (int i = static_cast<int>(pad_size) - 1; i >= 0; --i) {
        for (unsigned int j = 0; j < new_dimy; ++j) {
            for (unsigned int k = 0; k < new_dimz; ++k) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[(i + 1) + j * new_dimx + k * new_dimx * new_dimy];
            }
        }
    }

    // pad along +x
    for (unsigned int i = new_dimx - pad_size; i < new_dimx; ++i) {
        for (unsigned int j = 0; j < new_dimy; ++j) {
            for (unsigned int k = 0; k < new_dimz; ++k) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[(i - 1) + j * new_dimx + k * new_dimx * new_dimy];
            }
        }
    }

    // pad along -y
    for (int j = static_cast<int>(pad_size) - 1; j >= 0; --j) {
        for (unsigned int i = 0; i < new_dimx; ++i) {
            for (unsigned int k = 0; k < new_dimz; ++k) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[i + (j + 1) * new_dimx + k * new_dimx * new_dimy];
            }
        }
    }

    // pad along +y
    for (unsigned int j = new_dimy - pad_size; j < new_dimy; ++j) {
        for (unsigned int i = 0; i < new_dimx; ++i) {
            for (unsigned int k = 0; k < new_dimz; ++k) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[i + (j - 1) * new_dimx + k * new_dimx * new_dimy];
            }
        }
    }


    // pad along -z
    for (int k = static_cast<int>(pad_size) - 1; k >= 0; --k) {
        for (unsigned int i = 0; i < new_dimx; ++i) {
            for (unsigned int j = 0; j < new_dimy; ++j) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[i + j * new_dimx + (k + 1) * new_dimx * new_dimy];
            }
        }
    }

    // pad along +z
    for (unsigned int k = new_dimz - pad_size; k < new_dimz; ++k) {
        for (unsigned int i = 0; i < new_dimx; ++i) {
            for (unsigned int j = 0; j < new_dimy; ++j) {
                (*new_vol)[i + j * new_dimx + k * new_dimx * new_dimy] =
                    (*new_vol)[i + j * new_dimx + (k - 1) * new_dimx * new_dimy];
            }
        }
    }
}

/**
 * @brief      Create a 3-D gaussian filter kernel
 *
 * @param      filter  The gaussian fileter kernel
 * @param[in]  sizex   The x dimension
 * @param[in]  sizey   The y dimension
 * @param[in]  sizez   The z dimension
 * @param[in]  std     The standard deviation
 */
void gaussian_filter(float** filter, unsigned int sizex, unsigned int sizey,
                     unsigned int sizez, float std) {
    float sum = 0.0f;
    *filter = (float*)calloc(sizex * sizey * sizez, sizeof(float));

    for (unsigned int i = 0; i < sizex; ++i) {
        for (unsigned int j = 0; j < sizey; ++j) {
            for (unsigned int k = 0; k < sizez; ++k) {
                float x = (static_cast<float>(i) - (static_cast<float>(sizex) - 1.0f) / 2.0f);
                float y = (static_cast<float>(j) - (static_cast<float>(sizey) - 1.0f) / 2.0f);
                float z = (static_cast<float>(k) - (static_cast<float>(sizez) - 1.0f) / 2.0f);
                (*filter)[i + j * sizex + k * sizex * sizey] =
                    expf(-(x * x + y * y + z * z) / (2.0f * std * std));
                sum += (*filter)[i + j * sizex + k * sizex * sizey];
            }
        }
    }

    // normalization
    float factor = 1.0f / sum;

    for (unsigned int i = 0; i < sizex; ++i) {
        for (unsigned int j = 0; j < sizey; ++j) {
            for (unsigned int k = 0; k < sizez; ++k) {
                (*filter)[i + j * sizex + k * sizex * sizey] *= factor;
            }
        }
    }
}

/**
 * @brief      Initialize the lower label.
 *
 * @param      vol_new  The new volume
 * @param      vol      The old volume
 */
__global__ void init_lower_label(unsigned char* vol_new, unsigned int* vol) {
    unsigned int idx1d = flatten_3d_to_1d(blockIdx, gridDim);
    vol_new[idx1d * sizeof(unsigned int) + 0] = vol[idx1d] & MED_MASK; // bytes[7]
}

/**
 * @brief      Creates a binary mask for the specified label
 *
 * @param      vol          The volume
 * @param      binary_mask  The binary mask
 * @param[in]  label        The label
 */
__global__ void create_binary_mask(unsigned int* vol, float* binary_mask, unsigned int label) {
    unsigned int idx1d = flatten_3d_to_1d(blockIdx, gridDim);
    binary_mask[idx1d] = ((vol[idx1d] & MED_MASK) == label ? 1.0f : 0.0f);
}

/**
 * @brief      Perform gaussian smoothing
 *
 * @param      binary_mask  The binary mask
 * @param      mask         The mask after gaussian smoothing
 */
__global__ void gaussian_blur(float* binary_mask, float* mask) {
    unsigned int pad_size = MCX_SVMC_GKERNEL_SIZE / 2;
    float temp = 0.0f;

    for (unsigned int i = 0; i < MCX_SVMC_GKERNEL_SIZE; ++i) {
        for (unsigned int j = 0; j < MCX_SVMC_GKERNEL_SIZE; ++j) {
            for (unsigned int k = 0; k < MCX_SVMC_GKERNEL_SIZE; ++k) {
                temp += binary_mask[flatten_3d_to_1d(make_uint3(blockIdx.x + i, blockIdx.y + j, blockIdx.z + k),
                                                                                            make_uint3(gridDim.x + pad_size * 2, gridDim.y + pad_size * 2, gridDim.z + pad_size * 2))] *
                        gfilter[flatten_3d_to_1d(make_uint3(MCX_SVMC_GKERNEL_SIZE - 1 - i, MCX_SVMC_GKERNEL_SIZE - 1 - j, MCX_SVMC_GKERNEL_SIZE - 1 - k),
                                                                                                   make_uint3(MCX_SVMC_GKERNEL_SIZE, MCX_SVMC_GKERNEL_SIZE, MCX_SVMC_GKERNEL_SIZE))];
            }
        }
    }

    mask[flatten_3d_to_1d(blockIdx, gridDim)] = temp;
}

// Extract isosurface and get the new volume for svmc simulation
__global__ void split_voxel(float* scalar_field, unsigned char* new_vol, unsigned int label) {
    // grid idx3d
    uint3 cube_idx3d = blockIdx + make_uint3(1, 1, 1);

    // vol dimension
    uint3 vol_dim = gridDim + make_uint3(1, 1, 1);

    // get index of the polygon configurations (0 - 255)
    float cube_values[8];
    unsigned char cube_index = 0;

    for (unsigned int i = 0; i < 8; ++i) {
        cube_values[i] = scalar_field[flatten_3d_to_1d(blockIdx + cube_vertices_local[i], vol_dim)];

        if (cube_values[i] < MCX_SVMC_ISOVALUE) {
            cube_index |= (1 << i);
        }
    }

    // if the voxel does not need to be split, terminate
    if (cube_index == 0 || cube_index == 0xFF) {
        return;
    }

    // get intersections (in the local coordinate of the grid) between isosurface and cube edges
    uint16_t edge_intersection_mask = edge_intersections[cube_index];
    float3 isosurface_vertices[12];

    for (unsigned int i = 0; i < 12; ++i) {
        if (edge_intersection_mask & 1) {
            // mcx volume lower corner is [0 0 0] while svmc volume lower corner is [1 1 1].
            // It is because the medium type is defined at the grid vertices.
            isosurface_vertices[i] = interpolate(make_float3(cube_vertices_local[edge_vertices[i][0]]),
                                                 make_float3(cube_vertices_local[edge_vertices[i][1]]),
                                                 cube_values[edge_vertices[i][0]],
                                                 cube_values[edge_vertices[i][1]],
                                                 MCX_SVMC_ISOVALUE);
        }

        edge_intersection_mask >>= 1;
    }

    // get isosurface triangles (in the local coordinate of the grid)
    float3 isosurface_centroid = make_float3(0.0f, 0.0f, 0.0f);
    float3 isosurface_normal = make_float3(0.0f, 0.0f, 0.0f);
    float isosurface_area = 0.0f;

    for (unsigned int i = 0; triangle_vertices[cube_index][i] != -1; i += 3) {
        // get a triangle
        float3& A = isosurface_vertices[triangle_vertices[cube_index][i]];
        float3& B = isosurface_vertices[triangle_vertices[cube_index][i + 1]];
        float3& C = isosurface_vertices[triangle_vertices[cube_index][i + 2]];
        float3 AB_x_AC = cross(B - A, C - A);
        float triangle_area = 0.5f * length(AB_x_AC);
        float3 triangle_normal = 0.5f * AB_x_AC / triangle_area; // AB_x_AC / length(AB_x_AC)
        float3 triangle_centroid = 1.0f / 3.0f * (isosurface_vertices[triangle_vertices[cube_index][i]] +
                                   isosurface_vertices[triangle_vertices[cube_index][i + 1]] +
                                   isosurface_vertices[triangle_vertices[cube_index][i + 2]]);

        // compress triangle information if mulitple triangles are present
        isosurface_area += triangle_area;
        isosurface_normal += triangle_area * triangle_normal;
        isosurface_centroid += triangle_area * triangle_centroid;
    }

    isosurface_normal = -isosurface_normal / isosurface_area;
    isosurface_centroid /= isosurface_area;

    // 1D index of the current grid
    unsigned int idx1d = flatten_3d_to_1d(cube_idx3d, vol_dim);
    unsigned int vol_length = vol_dim.x * vol_dim.y * vol_dim.z;

    // check if we are processing for lower label or upper label
    unsigned int* temp = (unsigned int*)new_vol;

    if (temp[idx1d + vol_length]) { // if we have already init isosurface normal, bytes[3-0] must not be zero
        // update upper volume and return
        new_vol[idx1d * sizeof(unsigned int) + 1] = label; // bytes[6]
        // printf("Grid #[%u %u %u], [%u %u %u %u %u %u %u %u]\n", cube_idx3d.x + 1, cube_idx3d.y + 1, cube_idx3d.z + 1,
        //     new_vol[idx1d * sizeof(unsigned int) + 0],
        //     new_vol[idx1d * sizeof(unsigned int) + 1],
        //     new_vol[idx1d * sizeof(unsigned int) + 2],
        //     new_vol[idx1d * sizeof(unsigned int) + 3],
        //     new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 0],
        //     new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 1],
        //     new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 2],
        //     new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 3]);
        return;
    }

    // update lower label
    new_vol[idx1d * sizeof(unsigned int) + 0] = label; // bytes[7]

    // convert float vectors to gray-scale vectors (0-255) and update the new volume
    new_vol[idx1d * sizeof(unsigned int) + 2] = (unsigned char)(isosurface_centroid.x * 255.0f); // bytes[5]
    new_vol[idx1d * sizeof(unsigned int) + 3] = (unsigned char)(isosurface_centroid.y * 255.0f); // bytes[4]
    new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 0] = (unsigned char)(isosurface_centroid.z * 255.0f); // bytes[3]
    new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 1] = min((unsigned char)floorf((isosurface_normal.x + 1.0f) * 255.0f * 0.5f), 254); // bytes[2]
    new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 2] = min((unsigned char)floorf((isosurface_normal.y + 1.0f) * 255.0f * 0.5f), 254); // bytes[1]
    new_vol[(idx1d + vol_length) * sizeof(unsigned int) + 3] = min((unsigned char)floorf((isosurface_normal.z + 1.0f) * 255.0f * 0.5f), 254); // bytes[0]
}

/**
 * @brief      In a 3-D cartesian coordinate,
 *             given the isovalue, compute the interpolated point between two points.
 *
 * @param[in]  a         first point
 * @param[in]  b         second point
 * @param[in]  a_val     value of first point
 * @param[in]  b_val     value of second point
 * @param[in]  isovalue  value of interpolated point
 *
 * @return     Interpolated position
 */
__device__ float3 interpolate(float3 a, float3 b, float a_val, float b_val, float isovalue) {
    return b_val == a_val ? a : (b + (a - b) * ((isovalue - b_val) / (a_val - b_val)));
}

/**
 * @brief      Convert 3D index to 1D index
 *
 * @param[in]  idx3d  3-D index
 * @param[in]  dim    The dimension of the 3-D matrix
 *
 * @return     1-D index
 */
__device__ unsigned int flatten_3d_to_1d(uint3 idx3d, uint3 dim) {
    return idx3d.x + idx3d.y * dim.x + idx3d.z * dim.x * dim.y;
}